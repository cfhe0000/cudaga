#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "ind_ga.h"

#define DEBUG 0

__device__ int parseBits(unsigned char* bits, unsigned char* buffer);
__device__ void fillGeneRandom(unsigned char* bits, unsigned *seed);
__device__ int gameOver(char board[6][7], char column);


__host__ void print_complete(chromo *pool) {
	FILE *fp;
	fp = fopen("final_chromo.txt", "w");
	int i, j;
	int fitness;
	if (!fp) {
		printf("Error writing chromosome out\n");
		return;
	}

	i = 0;
	fitness = 0;
	for (j = 0; j < NUM_INDIVIDUALS; j++) {
		if (pool[j].fitness > fitness) {
			fitness = pool[j].fitness;
			i = j;
		}
	}

	for (j = 0; j < CHROMO_LENGTH; j++) {
		fprintf(fp, "%d %d \n", pool[i].bits[j][0],pool[i].bits[j][1]);
	}
	fprintf(fp, "\n");
	fclose(fp);


}

__device__ int create_individual(chromo *parents, chromo *child, unsigned *seed)
{
	int xpoint = (grand(seed)) * CHROMO_LENGTH;
	int i, x;
	if (!parents || !child)
		return -1;

	child->fitness = 0;

	for (i = 0; i < xpoint; i++) {
		child->bits[i][0] = parents[0].bits[i][0];
		child->bits[i][1] = parents[0].bits[i][1];
	}

	for (; i < GENE_BYTES; i++) {
		child->bits[i][0] = parents[1].bits[i][0];
		child->bits[i][1] = parents[1].bits[i][1];
	}

	for (x = 0; x < CHROMO_LENGTH; x++) {
		for (i = 0; i < (GENE_BYTES * 8); i++) {
			if ((grand(seed)) < MUTATION_RATE) {
				child->bits[x][i / 8] ^= (1 << (i % 8));
			}
		}
	}

	return 0;
}



__device__ int init_individual(chromo *ind, unsigned *seed)
{	
	if (!ind)
		return -1;
	int i, j;

	ind->fitness = 0;

	for (j = 0; j < CHROMO_LENGTH; j++) {
		for (i = 0; i < GENE_BYTES; i++) {
			ind->bits[j][i] = (grand(seed)) * 255;
		}
	}

	return 0;
}

__device__ void cpy_ind(chromo *ind, chromo *old)
{
	int i,j;
	ind->fitness = old->fitness;
	for (j = 0; j < CHROMO_LENGTH; j++) {
		for (i = 0; i < GENE_BYTES; i++) {
			ind->bits[j][i] = old->bits[j][i];
		}
	}

	return;
}

__device__ float gabs(float val) 
{
	if (val > 0) {
		return val;
	} else {
		return -val;
	}
}

__device__ int calc_fitness(chromo *players, unsigned *seed)
{
	char gamestate = 0; // 0 for ongoing, 1 for just won, 2 for tie

	// Player 1 is black, player 2 is red
	char theBoard[6][7] = { { 0 } }; // 0 is empty, 1 is black, 2 is red
	// The bottom row is row 0, the left column is col 0

	// The bottom cell in each column contains, in bits 4-2, the index
	// of the bottom most empty cell in that column (therefore if bits
	// 4-2 contain 6, the column is FULL

	char state[2] = { 0 };

	char lastMove[2] = { 0 };

	int fitness[2] = { 0 };

	char nextPlay;
	char turn = 0; // 0 for player1, 1 for player2

	char lowByte = 0;
	char hiByte = 0;

	char height = 0;
	char seq_illegal_turns = 0;

	if (!players)
		return -1;

	
	// This function overwrites players' previous
	// fitness, if any

	// First player one must go randomly
	nextPlay = (int) ((grand(seed) / (float) RAND_MAX) * 7);
	// This is the first play so don't check board bounds
	theBoard[0][nextPlay] = 1; // Black plays here
	lastMove[turn] = nextPlay;
	theBoard[0][nextPlay] = (((1 & 7) << 2) | (theBoard[0][nextPlay] & 3));
	turn ^= 1;

	gamestate = 0;
	while (!gamestate) {

		lowByte = players[turn].bits[((((int)state[turn] << 3) | lastMove[turn ^ 1])) & 0x07ff][0];
		hiByte = players[turn].bits[((((int)state[turn] << 3) | lastMove[turn ^ 1])) & 0x07ff][1];

		state[turn] = hiByte;
		nextPlay = (7 & lowByte);

		// Check for invalid move
		height = ((theBoard[0][nextPlay] & (7 << 2)) >> 2);
		lastMove[turn] = nextPlay;
		if (height > 5) {
			// Illegal move (nub)
			seq_illegal_turns++;
			fitness[turn] -= ILLEGAL_MOVE_PENALTY;
		} else { // Legal move, record it
			seq_illegal_turns = 0;
			fitness[turn] += LEGAL_MOVE_REWARD;
			// Update the bookkeeping
			theBoard[0][nextPlay] = ((height + 1) << 2) | (theBoard[0][nextPlay] & 3);
			// Place the token in the board
			if (turn) // Red played
				theBoard[height][nextPlay] = (theBoard[height][nextPlay] & ~3) | 2;
			else
				// Black played
				theBoard[height][nextPlay] = (theBoard[height][nextPlay] & ~3) | 1;

			gamestate = gameOver(theBoard, nextPlay);
		}

		//printBoard(theBoard);

		turn ^= 1;		// Change turns
		if (seq_illegal_turns == 100) {
			gamestate = 2;
			break;
		}
	}



	if (gamestate == 1) { // Someone just won
		fitness[turn ^ 1] += WIN_REWARD; // gamestate moved - switching rewards
		fitness[turn] -= LOSE_PENALTY;

#if DEBUG
		printf("%s won:\n", turn ? "Red":"Black");
#endif
	} else if (gamestate == 2) { // It was a tie
		fitness[0] += TIE_REWARD;
		fitness[1] += TIE_REWARD;
#if DEBUG
		printf("It was a tie:\n");
#endif
	}

#if DEBUG
	printBoard(theBoard);
#endif

	players[0].fitness += fitness[0];
	if (players[0].fitness < 1) players[0].fitness = 1;
	players[1].fitness += fitness[1];
	if (players[1].fitness < 1) players[1].fitness = 1;

	return 0;
}

__device__ 
int gameOver(char board[6][7], char column) {
	char col, row;
	char count;
	char height = ((board[0][column] & (7 << 2)) >> 2) - 1;

	// Check horizontally first
	count = 1;
	col = column;
	while (col > 0) {
		if ((board[height][col] & 3) == (board[height][col - 1] & 3)) {
			col--;
			count++;
		}
		else
			break;
	}
	col = column;
	while (col < 5) {
		if ((board[height][col] & 3) == (board[height][col + 1] & 3)) {
			col++;
			count++;
	}
		else
			break;
	}
	if (count >= 4) {
#if DEBUG
		printf("H Winning move: R%d C%d\n",height, column);
#endif
		return 1;	// This move won
	}

	//Check vertically next
	count = 1;
	row = height;
	if (height >= 3) {	// Need at least 4 checkers in this column
		while (row > 0) {
			if ((board[row][column] & 3) == (board[row - 1][column] & 3)) {
				row--;
				count++;
			}
			else
				break;
		}
		if (count >= 4) {
#if DEBUG
			printf("V Winning move: R%d C%d\n",height, column);
#endif
			return 1;	// This move won
		}
	}

	// Check for diagonal wins here
	// First check for  /  diagonals
	count = 1;
	col = column;
	row = height;
	while (row > 0 && col > 0) {		// check down,left first
		if ((board[row][col] & 3) == (board[row - 1][col - 1] & 3)) {
			row--;
			col--;
			count++;
		}
		else
			break;
	}
	col = column;
	row = height;
	while (row < 4 && col < 5) {		// check up, right next
		if ((board[row][col] & 3) == (board[row + 1][col + 1] & 3)) {
			row++;
			col++;
			count++;
		}
		else
			break;
	}
	if (count >= 4) {
#if DEBUG
		printf("D/ Winning move: R%d C%d\n",height, column);
#endif
		return 1;	// This move won
	}

	// Now check for  \  diagonals
	count = 1;
	col = column;
	row = height;
	while (row > 0 && col < 5) {		// check down,right first
		if ((board[row][col] & 3) == (board[row - 1][col + 1] & 3)) {
			row--;
			col++;
			count++;
		}
		else
			break;
	}
	col = column;
	row = height;
	while (row < 4 && col > 0) {		// check up, left next
		if ((board[row][col] & 3) == (board[row + 1][col - 1] & 3)) {
			row++;
			col--;
			count++;
		}
		else
			break;
	}
	if (count >= 4) {
#if DEBUG
		printf("D\\ Winning move: R%d C%d\n",height, column);
#endif
		return 1;	// This move won
	}

	// Check for a tie (full board)
	count = 1;
	for (col = 0; col < 7; col++) {
		height = ((board[0][col] & (7 << 2)) >> 2);
		if (height < 6) {
			count = 0;
			break;
		}
	}
	if (count) {
#if DEBUG
		printf("Tieing move: R%d C%d\n",height, column);
#endif
		return 2;	// It was a tie!
	}
	return 0;

}



void printBoard(char board[6][7]) {
	int row, col;
	for (row = 5; row >= 0; row--) {
		printf("|");
		for (col = 0; col < 7; col++) {
			if ((board[row][col] & 3) == 0)
				printf(" |");
			else if ((board[row][col] & 3) == 1)
				printf("x|");
			else if ((board[row][col] & 3) == 2)
				printf("o|");
			else
				printf("?|");
		}
		printf("\n");
	}
	printf(" -+-+-+-+-+-+- \n\n");
}
