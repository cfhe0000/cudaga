#include "hip/hip_runtime.h"
#include "cuda_ga.h"
#include "random.h"
#include <sm_11_atomic_functions.h>

//this runs the main loop of all the threads which will call insert_roulette, init_individual, and 
//calc_fitness until a solution is found or the max number of “generations” is reached for one 
//of the threads.
//this function assumes that pool has sizeof(chromo) * POOL_SIZE bytes allocated in gpu
//memory.
//when this returns the last “generation” of individuals will be stored in the pool in order of fitness
//best first.
__global__ void run_ga(mutex *lock, chromo *pool, unsigned *seeds)
{
	chromo locals[NUM_OFFSPRING];
	chromo parents[2];
	int i,j;
	int th_id = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned seed = seeds[th_id];

	if (th_id == 0) {
		for (i = 0; i < NUM_INDIVIDUALS; i++) {
			init_individual(&pool[i], &seed);
		}
		lock[1] = 0;
	} else {
		while (lock[1]);
	}
	mutex_lock(lock, &seed);
	//for (i = 0; i < NUM_OFFSPRING; i++) {
	//	cpy_ind(&locals[i], &pool[i + th_id * NUM_OFFSPRING]);
	//}
	mutex_unlock(lock);
	return;
	for (i = 0; i < (NUM_OFFSPRING - 1); i += 2) {
		calc_fitness(&locals[i], &locals[i + 1]);
	}
	if (NUM_OFFSPRING & 0x01) {
		calc_fitness(&locals[0], &locals[NUM_OFFSPRING - 1]);
	}

	for (i = 0; (i < MAX_GENERATIONS); i++) {
		mutex_lock(lock, &seed);
		insert_roulette(lock, pool, locals, parents, &seed);
		if (lock[1]) {
			seeds[th_id] = seed;
			mutex_unlock(lock);
			return;
		}
		for (j = 0; j < NUM_OFFSPRING; j++) {
			if (locals[j].fitness >= END_FITNESS) {
				lock[1] = 1;
				seeds[th_id] = seed;
				mutex_unlock(lock);
				return;
			}
		}
		mutex_unlock(lock);

		for (j = 0; j < NUM_OFFSPRING; j++) {
			create_individual(parents, &locals[i], &seed);
		}
		for (j = 0; j < NUM_OFFSPRING; j += 2) {
			calc_fitness(&locals[i], &locals[i + 1]);
		}
		if (NUM_OFFSPRING & 0x01) {
			calc_fitness(&locals[0], &locals[NUM_OFFSPRING - 1]);
		}
	}

	return;
}

//locks the mutex, then inserts the new individuals to the pool if fit enough
//then selects new individuals using roulette wheel, storing in locals, then unlocks the mutex
//number of individuals inserted into pool is based on NUM_OFFSPRING
__device__ int insert_roulette(mutex *lock, chromo *pool, chromo *locals, 
									chromo *parents, unsigned *seed)
{
    int fitness_sum;

    fitness_sum = insert(pool, locals);

    roulette(pool, parents, fitness_sum, seed);

	return 0;
}

//this is called by insert_roulette, good for code separation
__device__ int insert(chromo *pool, chromo *locals)
{
    signed int i,j,k;
    int fit_sum = 0;
    int worst[NUM_OFFSPRING];
    int flag;

    for (i = 0; i < NUM_OFFSPRING; i++) {
        worst[i] = 250000;
    }

    for (j = 0; j < NUM_OFFSPRING; j++) {
        fit_sum = 0;
        for (i = 0; i < NUM_INDIVIDUALS; i++) {
            fit_sum += pool[i].fitness;

            if ((worst[j] == 250000) ||
                    (pool[i].fitness < pool[worst[j]].fitness)) {
                flag = 0;
                for (k = j - 1; k >= 0; k--) {
                    if (worst[k] == i) {
                        flag = 1;
                        break;
                    }
                }

                if (!flag) {
                    worst[j] = i;
                }
            }
        }
    }


    for (i = 0; i < NUM_OFFSPRING; i++) {
        for (j = NUM_OFFSPRING; j > 0; j--) {
            if ((pool[worst[j - 1]].fitness < locals[i].fitness)) break;
        }

        if (!j) continue;
        j--;
        for (k = 0; k < j; k++) {
            cpy_ind(&pool[worst[k]], &pool[worst[k + 1]]);
        }

        fit_sum -= pool[worst[j]].fitness;
        cpy_ind(&pool[worst[j]], &locals[i]);
        fit_sum += pool[worst[j]].fitness;
    }

	return fit_sum;
}

//this is called by insert_roulette, good for code separation
__device__ int roulette(chromo *pool, chromo *parents, int sum, unsigned *seed)
{
	int rand_val = (grand(seed) * sum);
    int total_fit = 0;
	int i;

    for (i = 0; i < NUM_INDIVIDUALS - 1; i++) {
        total_fit += pool[i].fitness;
        if (total_fit > rand_val) break;
    }

    cpy_ind(&parents[0], &pool[i]);
    rand_val = (grand(seed) * sum);
    total_fit = 0;

    for (i = 0; i < NUM_INDIVIDUALS - 1; i++) {
        total_fit += pool[i].fitness;
        if (total_fit > rand_val) break;
    }

    cpy_ind(&parents[1], &pool[i]);

    return 0;
}

#include "mutex_testing/mutex_testing.cu"
#include "random.cu"
#include "ind_ga.cu"
#include "main.c"
